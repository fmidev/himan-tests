#include "cuda_helper.h"
#include "himan_common.h"
#include "numerical_functions.h"
#include "timer.h"

#include "numerical_functions_helper.h"

#include <boost/test/unit_test.hpp>

#define BOOST_TEST_MODULE numerical_functions_cuda

using namespace std;
using namespace himan;

BOOST_AUTO_TEST_CASE(FILTER2DGPU_CMP_CPU_TIMING)
{
	for (auto f : vector<std::string>({"Filter2D", "Max2D"}))
	{
		for (auto h : vector<int>({3, 7, 21}))
		{
			for (auto i : vector<int>({1, 5, 10, 15, 20, 40, 50}))
			{
				// Filter a plane with given filter kernel with CUDA
				const int M = i * 100;
				const int N = i * 50;
				himan::matrix<double> A(M, N, 1, MissingDouble());  // input
				himan::matrix<double> B(h, h, 1, MissingDouble());  // convolution kernel
				himan::matrix<double> D(M, N, 1, MissingDouble());  // reference for testing

				FilterTestSetup(A, B, D);

				himan::timer CPUTimer;
				himan::timer GPUTimer;
				double* d = 0;
				hipMalloc(&d, 1);
				hipFree(d);
				// Compute the cpu version

				if (f == "Filter2D")
				{
					CPUTimer.Start();
					himan::matrix<double> cpuResult = numerical_functions::Filter2D(A, B, false);
					CPUTimer.Stop();
					GPUTimer.Start();
					himan::matrix<double> gpuResult = numerical_functions::Filter2D(A, B, true);
					GPUTimer.Stop();
				}
				else if (f == "Max2D")
				{
					CPUTimer.Start();
					himan::matrix<double> cpuResult = numerical_functions::Max2D(A, B, false);
					CPUTimer.Stop();
					GPUTimer.Start();
					himan::matrix<double> gpuResult = numerical_functions::Max2D(A, B, true);
					GPUTimer.Stop();
				}

				std::cout << f << " input matrix (" << A.SizeX() << "x" << A.SizeY() << ") stencil size (" << B.SizeX()
				          << "x" << B.SizeY() << ") CPU: " << CPUTimer.GetTime() << " ms"
				          << " GPU: " << GPUTimer.GetTime() << " ms" << std::endl;

				if (i > 5)
				{
					BOOST_REQUIRE(CPUTimer.GetTime() > GPUTimer.GetTime());
				}
			}
		}
	}
}
