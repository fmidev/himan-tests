#include "cuda_helper.h"
#include "himan_common.h"
#include "numerical_functions.h"
#include "timer.h"

#include "numerical_functions_helper.h"

#include <boost/test/unit_test.hpp>

#define BOOST_TEST_MODULE numerical_functions_cuda

using namespace std;
using namespace himan;

const double kEpsilon = 1e-9;

const bool USE_CUDA = []() -> bool {
	int devCount;
	hipError_t err = hipGetDeviceCount(&devCount);
	return (err == hipSuccess);
}();

BOOST_AUTO_TEST_CASE(FILTER2DCUDA_SMALL)
{
	if (!USE_CUDA)
	{
		std::cerr << "Cuda device not found\n";
		return;
	}

	// Filter a plane with given filter kernel with CUDA
	himan::matrix<double> A(11, 8, 1, MissingDouble());  // input
	himan::matrix<double> B(3, 3, 1, MissingDouble());   // convolution kernel
	himan::matrix<double> D(11, 8, 1, MissingDouble());  // reference for testing

	FilterTestSetup(A, B, D);

	// Compute smoothened matrix with CUDA
	auto C = numerical_functions::Reduce2DGPU(
	    A, B, [ =, *this ] __device__(double& val1, double& val2, const double& a, const double& b) {
		    if (IsValid(a * b))
		    {
			    val1 += a * b;
			    val2 += b;
		    }
	    },
	    [ =, *this ] __device__(const double& val1, const double& val2) {
		    return val2 == 0.0 ? MissingDouble() : val1 / val2;
	    },
	    0.0, 0.0);

	// Compare results
	for (size_t i = 0; i < C.Size(); ++i)
	{
		BOOST_CHECK_CLOSE(C.At(i), D.At(i), kEpsilon);
	}

	// computed filtered matrix
	std::cout << "Matrix C computed with Filter2D:" << std::endl;
	for (size_t i = 0; i < C.SizeX(); ++i)
	{
		for (size_t j = 0; j < C.SizeY(); ++j)
		{
			std::cout << C.At(i, j, 0) << " ";
		}
		std::cout << std::endl;
	}

	std::cout << std::endl << "Matrix D as reference case for Filter2D computation:" << std::endl;

	for (size_t i = 0; i < D.SizeX(); ++i)
	{
		for (size_t j = 0; j < D.SizeY(); ++j)
		{
			std::cout << D.At(i, j, 0) << " ";
		}
		std::cout << std::endl;
	}
}

// Compare against the CPU version
BOOST_AUTO_TEST_CASE(FILTER2DCUDA_LARGE_CMP_CPU)
{
	if (!USE_CUDA)
	{
		std::cerr << "Cuda device not found\n";
		return;
	}

	// Filter a plane with given filter kernel with CUDA
	himan::matrix<double> A(807, 301, 1, MissingDouble());  // input
	himan::matrix<double> B(3, 3, 1, MissingDouble());      // convolution kernel
	himan::matrix<double> D(807, 301, 1, MissingDouble());  // reference for testing

	FilterTestSetup(A, B, D);

	himan::timer CPUTimer;
	himan::timer GPUTimer;
	double* d = 0;
	hipMalloc(&d, 1);
	hipFree(d);
	// Compute the cpu version

	CPUTimer.Start();
	himan::matrix<double> cpuResult = numerical_functions::Filter2D<double>(A, B, false);
	CPUTimer.Stop();

	GPUTimer.Start();
	himan::matrix<double> C = numerical_functions::Filter2DGPU<double>(A, B);
	GPUTimer.Stop();

	// Compare results
	for (size_t i = 0; i < C.Size(); ++i)
	{
		BOOST_CHECK_CLOSE(C.At(i), D.At(i), kEpsilon);
	}

	for (size_t i = 0; i < C.Size(); ++i)
	{
		BOOST_CHECK_CLOSE(C.At(i), cpuResult.At(i), kEpsilon);
	}

	std::cout << "Filter2D(CPU) time for input matrix (" << A.SizeX() << "x" << A.SizeY() << "): " << CPUTimer.GetTime()
	          << " ms" << std::endl;
	std::cout << "Filter2D(GPU) time for input matrix (" << A.SizeX() << "x" << A.SizeY() << "): " << GPUTimer.GetTime()
	          << " ms" << std::endl;
}
